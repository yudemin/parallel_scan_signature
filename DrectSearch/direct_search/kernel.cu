#include "hip/hip_runtime.h"
﻿#include <Windows.h>
#include <iostream>
#include <fstream>
#include <fstream>
#include <vector>
#include <chrono> 
#include <iomanip> 
#include <mutex>
#include <string.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

using namespace std;

__global__ void scan(unsigned char* byte_file, unsigned char* sign, int part_size, int sign_size, int file_size, int threads_per_block, char* mask)
{
    int thread = blockIdx.x* threads_per_block* part_size; // номер текущего потока
    int i = threadIdx.x;
    int offset = thread + (i * part_size); // вычисление сдвига по файлу

    int sign_shift = 0;
    while ((sign_shift < part_size) && (offset + sign_shift < file_size - sign_size))  { // цикл алгоритма с учетом перекрытий
        bool sign_check = true;
        for (int i = 0; i < sign_size; i++)
        {                      
            if (byte_file[offset + sign_shift + i] != sign[i])
            {                                
                if (mask[i] == '1') // проверяем, является ли байт ключевым
                {                  
                    sign_check = false;
                    break;
                }
                else
                {
                    continue;
                }
            }            
        }
        if (sign_check == true)
        {
            /* сигнатура найдена */
            //printf("sign found on address:  %04x (block: %d, tread: %d)\n", (offset + sign_shift), blockIdx.x, threadIdx.x);
        }
        sign_shift++;
    }
}

int main()
{
    printf("direct search algoritm on GPU\n");
    /* процесс считывания данных */
    const char* filename = "D:/_Politech/ScanSignature/RealtekHDAudio_Rus_Setup.exe"; // используется EXE звукового драйвера, т.к. самый большой найденный EXE
    ifstream file; // файл открывается для бинарного считывания 
    file.open(filename, ios_base::binary);
    file.seekg(0, ios::end); // определение длины файла
    size_t fileSize = file.tellg();
    file.seekg(0, ios::beg);
    vector<byte> byte_file(fileSize, 0); // создание вектора, в котором будут хранится все байты из файла
    file.read(reinterpret_cast<char*>(&byte_file[0]), fileSize); // считывание  
    file.close(); // файл закрывается

    vector<unsigned char> sign_buffer = { 0x80, 0x3E, 0x00, 0x0F }; // объявление сигнатуры 
    char mask_[20] = "1111100000000000"; // объявление маски
    int sign_size = sign_buffer.size(); 
    int file_size = byte_file.size();


    auto start = chrono::high_resolution_clock::now(); // замер времени для инициализации
    /* запись файла в память ГПУ */
    unsigned char* bytefile;
    hipMallocManaged(&bytefile, file_size * sizeof(char));
    for (int i = 0; i < file_size; i++) {
        bytefile[i] = byte_file[i];
    }

    /* запись сигнатуры в память ГПУ */
    unsigned char* sign;
    hipMallocManaged(&sign, sign_size * sizeof(char));
    for (int i = 0; i < sign_size; i++) {
        sign[i] = sign_buffer[i];
    }

    /* запись маски в память ГПУ */
    char* mask;
    hipMallocManaged(&mask, sign_size * sizeof(char));
    for (int i = 0; i < sign_size; i++) {
        mask[i] = mask_[i];
    }

    auto stop = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::milliseconds>(stop - start);
    printf("sign: %s\nmask: %s\n", sign, mask);
    printf("initialising time: %d milliseconds.\n", duration.count());

    /* инициализация параметров */
    int treads[5] = { 256, 512, 1024, 2048, 4096}; // массив с количеством потоков для каждого выполнения бенчмарка
    int threads_ = 0;
    int blocks = 16;
    for (int i = 0; i < 5; i++) 
    {
        threads_ = treads[i];
        int part_size = file_size / threads_;
        int threads_per_block = threads_ / blocks;

        auto start = chrono::high_resolution_clock::now();
        scan <<<blocks, threads_per_block >>> (bytefile, sign, part_size, sign_size, file_size, threads_per_block, mask); // вызов функции на GPU 
        hipDeviceSynchronize(); // ожидание завершения всех потоков
        auto stop = chrono::high_resolution_clock::now();
        auto duration = chrono::duration_cast<chrono::milliseconds>(stop - start);
        printf("for %d blocks and %d treads execution time: %d milliseconds.\n", blocks, threads_, duration.count());
    }
    /* освобождение памяти */
    hipFree(bytefile);
    hipFree(sign);
    hipFree(mask);
    cout << "\npress enter for exit";
    cin.ignore();    
}

